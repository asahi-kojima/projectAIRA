#include "hip/hip_runtime.h"

#include "Affine.h"
#include <random>

namespace
{


	__global__ void affine_forward_gpu_impl(
		f32* y, 
		f32* x, 
		f32* A,
		f32* b, 
		u32 batchSize, u32 outputSize, u32 inputSize)
	{
		u32 xid = blockIdx.x * blockDim.x + threadIdx.x;
		u32 yid = blockIdx.y * blockDim.y + threadIdx.y;
		if (xid >= outputSize || yid >= batchSize)
		{
			return;
		}
		u32 id = yid * outputSize + xid;
		f32 result = 0.0f;
		for (u32 i = 0; i < inputSize; i++)
		{
			result += A[xid * inputSize + i] * x[yid * inputSize + i];
		}
		y[id] = result + b[xid];
	}

	__global__ void affine_backward_gpu_impl_input(DataType* dOut, DataType* dIn, DataType* A, u32 batchSize, u32 outputSize, u32 inputSize)
	{
		u32 xid = blockIdx.x * blockDim.x + threadIdx.x;//input
		u32 yid = blockIdx.y * blockDim.y + threadIdx.y;//batch

		if (xid >= inputSize || yid >= batchSize)
		{
			return;
		}

		f32 result = 0.0f;
		for (u32 i = 0; i < outputSize; i++)
		{
#if INDEX_DEBUG
			if (i * inputSize + xid >= outputSize * inputSize)
			{
				assert(0);
			}
			if (yid * outputSize + i >= batchSize * outputSize)
			{
				assert(0);
			}
#endif
			result += A[i * inputSize + xid] * dIn[yid * outputSize + i];
		}
		dOut[yid * inputSize + xid] = result;
	}

	__global__ void affine_backward_gpu_impl_weight(DataType* dA, DataType* dout, DataType* input, u32 batchSize, u32 outputSize, u32 inputSize)
	{
		u32 xid = blockIdx.x * blockDim.x + threadIdx.x;
		u32 yid = blockIdx.y * blockDim.y + threadIdx.y;
		if (xid >= inputSize || yid >= outputSize)
		{
			return;
		}

		u32 id = yid * inputSize + xid;

		f32 result = 0.0f;
		for (u32 N = 0; N < batchSize; N++)
		{
#if INDEX_DEBUG
			if (N * inputSize + xid >= batchSize * inputSize)
			{
				assert(0);
			}
			if (N * outputSize + yid >= batchSize * outputSize)
			{
				assert(0);
			}
#endif
			result += dout[N * outputSize + yid] * input[N * inputSize + xid];
		}

		dA[id] = result;
	}

	__global__ void affine_backward_gpu_impl_bias(DataType* dBias, DataType* dout, u32 batchSize, u32 outputSize)
	{
		u32 id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= outputSize)
		{
			return;
		}
		f32 result = 0.0f;
		for (u32 N = 0; N < batchSize; N++)
		{
#if INDEX_DEBUG
			if ((N * outputSize + id) >= batchSize * outputSize)
			{
				assert(0);
			}
#endif
			result += dout[N * outputSize + id];
		}
#if INDEX_DEBUG
		if (id >= outputSize)
		{
			assert(0);
		}
#endif
		dBias[id] = result;
	}
}

using AffineCore = aoba::nn::layer::AffineCore;
using LayerCore = aoba::nn::layer::LayerCore;


Layer Affine(u32 output_size)
{
	Layer affine = aoba::nn::layer::gen<AffineCore>("Affine", output_size);
	return affine;
}




AffineCore::AffineCore(u32 output_size)
	:LayerCore(1, 1, 1, 2)
	, m_batch_size(0)
	, m_input_size(0)
	, m_output_size(output_size)
{
}

AffineCore::~AffineCore()
{}

LayerCore::iotype  AffineCore::forward(const LayerCore::iotype& input_tensors)
{
	const auto& input_tensorcore = *getTensorCoreFrom(input_tensors[0]);

	auto dataSize_input = input_tensorcore.mDataSize;

	if (!m_init_finish)
	{
		m_batch_size = input_tensorcore.mBatchSize;
		m_input_size = input_tensorcore.mCHW;

		auto& child_tensorcore = m_child_tensorcore_tbl[0];
		child_tensorcore = std::make_shared<TensorCore>(input_tensorcore.mBatchSize, m_output_size, true);
		child_tensorcore->regist_parent_layercore(shared_from_this());

		auto& weight = m_parameter_tbl[0];
		auto& bias = m_parameter_tbl[1];
		weight = std::make_shared<TensorCore>(m_output_size, input_tensorcore.mCHW, true);
		bias = std::make_shared<TensorCore>(m_output_size, true);

		{
			std::random_device seed_gen;
			std::default_random_engine engine(seed_gen());
			std::normal_distribution<> dist(0.0f, std::sqrt(2.0f / m_input_size));
			for (u32 i = 0; i < weight->mDataSize; i++)
			{
				weight->_m_cpu_data_address[i] = affineWeight * static_cast<DataType>(dist(engine));
			}
			for (u32 i = 0; i < bias->mDataSize; i++)
			{
				bias->_m_cpu_data_address[i] = 0.0f;
			}
		}

		if (input_tensorcore._m_on_cuda)
		{
			m_on_cuda = true;

			child_tensorcore->to_cuda("");

			//�����p�����[�^��CUDA�ɑ���B
			weight->to_cuda("");
			bias->to_cuda("");
		}
		m_init_finish = true;
	}

	{
		if (input_tensorcore.mBatchSize != m_batch_size)
		{
			std::cout << "input batch size does not match" << std::endl;
			exit(1);
		}
		if (input_tensorcore.mCHW != m_input_size)
		{
			std::cout << "input chw size does not match" << std::endl;
			exit(1);
		}
	}

	{
		const auto& child_tensorcore = *m_child_tensorcore_tbl[0];
		const auto& weight = *m_parameter_tbl[0];
		const auto& bias = *m_parameter_tbl[1];


		//std::cout << "Affine forward " << (m_on_cuda ? "On GPU" : "on CPU") << std::endl;
		if (m_on_cuda)
		{
			auto output_address = child_tensorcore._m_gpu_data_address;
			auto input_address = input_tensorcore._m_gpu_data_address;
			auto weight_address = weight._m_gpu_data_address;
			auto bias_address = bias._m_gpu_data_address;

			dim3 block(32, 32);
			dim3 grid((m_output_size + block.x - 1) / block.x, (m_batch_size + block.y - 1) / block.y);
			affine_forward_gpu_impl << <grid, block >> > (
				output_address,
				input_address,
				weight_address,
				bias_address,
				m_batch_size,
				m_output_size,
				m_input_size);
			CUDA_SYNCHRONIZE_DEBUG;
		}
		else
		{
			affine_forward_cpu_impl(input_tensors);
		}
	}


	return iotype{ Tensor(m_child_tensorcore_tbl[0]) };
}


void AffineCore::backward()
{
	//std::cout << "Affine backward" << std::endl;
	if (std::shared_ptr<TensorCore> input_tensorcore = mInputTensorCoreTbl[0].lock())
	{
		auto dataSize = m_child_tensorcore_tbl[0]->mDataSize;
		auto output_grad_address = m_child_tensorcore_tbl[0]->_m_gpu_grad_data_address;
		auto input_address = input_tensorcore->_m_gpu_data_address;
		auto input_grad_address = input_tensorcore->_m_gpu_grad_data_address;
		auto weight_grad_address = m_parameter_tbl[0]->_m_gpu_grad_data_address;
		auto bias_grad_address = m_parameter_tbl[1]->_m_gpu_grad_data_address;

		//�p�����[�^�̋t�`��
		{
			if (m_on_cuda)
			{

				//Weight
				{
					dim3 block(16, 16);
					dim3 grid((m_input_size + block.x - 1) / block.x, (m_output_size + block.y - 1) / block.y);
					affine_backward_gpu_impl_weight << <grid, block >> > (
						weight_grad_address,
						output_grad_address,
						input_address,
						m_batch_size,
						m_output_size,
						m_input_size);
				}

				//Bias
				{
					dim3 block(16);
					dim3 grid((m_output_size + block.x - 1) / block.x);
					affine_backward_gpu_impl_bias << <grid, block >> >(
						bias_grad_address,
						output_grad_address,
						m_batch_size,
						m_output_size);
				}
				CUDA_SYNCHRONIZE_DEBUG;
			}
			else
			{
				affine_backward_cpu_impl_parameter(input_tensorcore);
			}
		}

		if (input_tensorcore->_m_need_grad)//���z�s�v�̏ꍇ�A�t�`���̓X�L�b�v�o����B
		{
			auto dataSize = m_child_tensorcore_tbl[0]->mDataSize;
			if (m_on_cuda)
			{
				dim3 block(16, 16);
				dim3 grid((m_input_size + block.x - 1) / block.x, (m_batch_size + block.y - 1) / block.y);
				affine_backward_gpu_impl_input << <grid, block >> > (
					input_grad_address,
					output_grad_address,
					weight_grad_address,
					m_batch_size,
					m_output_size,
					m_input_size);
				CUDA_SYNCHRONIZE_DEBUG;
			}
			else
			{
				affine_backward_cpu_impl_input(input_tensorcore);
			}
		}
	}
	else
	{
		std::cout << "Resource0 Error@ReLUCore::backward" << std::endl;
		exit(1);
	}
}



void AffineCore::affine_forward_cpu_impl(const LayerCore::iotype& input_tensors)
{
	const auto& input = *getTensorCoreFrom(input_tensors[0]);
	auto& output = *m_child_tensorcore_tbl[0];
	const auto& weight = *m_parameter_tbl[0];
	const auto& bias = *m_parameter_tbl[1];

	for (u32 N = 0, end = output.mBatchSize; N < end; N++)
	{
		for (u32 O = 0; O < m_output_size; O++)
		{
			u32 index = N * m_output_size + O;
			DataType result = 0.0f;
			for (u32 I = 0; I < m_input_size; I++)
			{
				result += weight(O, I) * input(N, I);
			}

			output(N, O) = result + bias(O);
		}
	}
}

void AffineCore::affine_backward_cpu_impl_input(const std::shared_ptr<TensorCore>& input_tensorcore)
{
	const auto& output = *m_child_tensorcore_tbl[0];
	auto& input = *input_tensorcore;
	auto& weight = *m_parameter_tbl[0];
	auto& bias = *m_parameter_tbl[1];

	for (u32 N = 0; N < m_batch_size; N++)
	{
		for (u32 I = 0; I < m_input_size; I++)
		{
			DataType result = 0.0f;
			for (u32 O = 0; O < m_output_size; O++)
			{
				result += weight(O, I) * output.d(N, O);
			}
			input.d(N, I) = result;
		}
	}
}

void AffineCore::affine_backward_cpu_impl_parameter(const std::shared_ptr<TensorCore>& input_tensorcore)
{
	const auto& output = *m_child_tensorcore_tbl[0];
	auto& input = *input_tensorcore;
	auto& weight = *m_parameter_tbl[0];
	auto& bias = *m_parameter_tbl[1];

	for (u32 O = 0; O < m_output_size; O++)
	{
		for (u32 I = 0; I < m_input_size; I++)
		{
			DataType result = 0.0f;
			for (u32 N = 0; N < m_batch_size; N++)
			{
				result += input(N, I) * output.d(N, O);
			}
			weight.d(O, I) = result;
		}

		DataType result = 0.0f;
		for (u32 N = 0; N < m_batch_size; N++)
		{
			result += output.d(N, O);
		}
		bias.d(O) = result;
	}
}