#include "hip/hip_runtime.h"
#include <random>
#include "Affine.h"
#include "nnLayer.h"
namespace
{


	__global__ void affine_forward_gpu_impl(
		f32* y,
		f32* x,
		f32* A,
		f32* b,
		u32 batchSize,
		u32 outputSize,
		u32 inputSize)
	{
		u32 O = blockIdx.x * blockDim.x + threadIdx.x;
		u32 N = blockIdx.y * blockDim.y + threadIdx.y;
		if (O >= outputSize || N >= batchSize)
		{
			return;
		}

		u32 index = N * outputSize + O;

		f32 result = 0.0f;
		for (u32 I = 0; I < inputSize; I++)
		{
			result += A[O * inputSize + I] * x[N * inputSize + I];
		}

		y[index] = result + b[O];
	}

	__global__ void affine_backward_gpu_impl_input(
		DataType* dOut,
		DataType* dIn,
		DataType* A,
		u32 batchSize, u32 outputSize, u32 inputSize)
	{
		u32 I = blockIdx.x * blockDim.x + threadIdx.x;//input
		u32 N = blockIdx.y * blockDim.y + threadIdx.y;//batch

		if (I >= inputSize || N >= batchSize)
		{
			return;
		}

		DataType result = 0.0f;
		for (u32 O = 0; O < outputSize; O++)
		{
#ifdef _DEBUG
			if (O * inputSize + I >= outputSize * inputSize)
			{
				assert(0);
			}
			if (N * outputSize + O >= batchSize * outputSize)
			{
				assert(0);
			}
#endif
			result += A[O * inputSize + I] * dIn[N * outputSize + O];
			//printf("A[%d, %d] = %f\n", O, I, A[O * inputSize + I]);
			//printf("DI[%d, %d] = %f\n", N, O, dIn[N * outputSize + O]);
		}
		dOut[N * inputSize + I] = result;
	}

	//Weight�p�����[�^
	__global__ void affine_backward_gpu_impl_weight(
		DataType* dA,
		DataType* dout,
		DataType* input,
		u32 batchSize,
		u32 outputSize,
		u32 inputSize)
	{
		u32 I = blockIdx.x * blockDim.x + threadIdx.x;
		u32 O = blockIdx.y * blockDim.y + threadIdx.y;
		if (I >= inputSize || O >= outputSize)
		{
			return;
		}

		u32 id = O * inputSize + I;

		DataType result = 0.0f;
		for (u32 N = 0; N < batchSize; N++)
		{
#if INDEX_DEBUG
			if (N * inputSize + I >= batchSize * inputSize)
			{
				assert(0);
			}
			if (N * outputSize + O >= batchSize * outputSize)
			{
				assert(0);
			}
#endif
			result += dout[N * outputSize + O] * input[N * inputSize + I];
		}

		dA[id] = result;
	}

	//Bias�p�����[�^
	__global__ void affine_backward_gpu_impl_bias(
		DataType* dBias,
		DataType* output_grad,
		u32 batchSize,
		u32 outputSize)
	{
		u32 O = blockIdx.x * blockDim.x + threadIdx.x;
		if (O >= outputSize)
		{
			return;
		}

		DataType result = 0.0f;
		for (u32 N = 0; N < batchSize; N++)
		{
#if INDEX_DEBUG
			if ((N * outputSize + O) >= batchSize * outputSize)
			{
				assert(0);
			}
#endif
			result += output_grad[N * outputSize + O];
		}
#if INDEX_DEBUG
		if (O >= outputSize)
		{
			assert(0);
		}
#endif
		dBias[O] = result;
	}
}

using namespace aoba::nn::layer;
using AffineCore = Layer::AffineCore;
using LayerSkeleton = Layer::LayerSkeleton;


Layer::nnLayer aoba::nn::layer::Affine(u32 output_size)
{
	Layer::nnLayer affine = gen<AffineCore>("Affine", output_size);
	return affine;
}




AffineCore::AffineCore(u32 output_size, DataType affineWeight)
	:LayerSkeleton(1, 1, 1, 2)
	, mAffineWeight(affineWeight)
	, m_output_size(output_size)
	, m_batch_size(0)
	, m_input_size(0)
	, mOutput(*m_output_tensorcore_tbl[0])
	, mWeight(*mTrainableParameterTbl[0])
	, mBias(*mTrainableParameterTbl[1])
{
}

AffineCore::~AffineCore()
{}

LayerSkeleton::iotype  AffineCore::forward(const LayerSkeleton::iotype& input_tensors)
{
	if (!m_init_finish)
	{
		initialize();
	}


	const auto& input = *getTensorCoreFrom(input_tensors[0]);

	const u32  input_batchSize = input.mBatchSize;
	const u32  input_chw = input.mCHW;
	const bool input_on_cuda = input.m_on_cuda;


	//�o�̓e���\���ƌP���p�����[�^�̌`��m�F���Ή�
	{
		m_batch_size = input_batchSize;
		m_input_size = input_chw;
		//m_on_cuda = input_on_cuda;

		//�o�̓e���\���̌`��ύX
		mOutput.reshapeAs(input_batchSize, m_output_size, input_on_cuda);

		//weight�̌`��ύX
		bool isWeightInit = mWeight.reshapeAs(m_output_size, m_input_size, input_on_cuda);

		//bias�̌`��ύX
		//�o�C�A�X�͏��񂾂�����������A����ȍ~�͕ω����Ȃ��B
		bool isBiasInit = mBias.reshapeAs(m_output_size, input_on_cuda);

		

		if (isWeightInit)
		{
#ifdef _DEBUG
			std::cout << "Weight Param was initialized." << std::endl;
#endif // _DEBUG
			std::random_device seed_gen;
			std::default_random_engine engine(seed_gen());
			std::normal_distribution<> dist(0.0f, std::sqrt(2.0f / m_input_size));
			for (u32 i = 0, end = mWeight.mDataSize; i < end; i++)
			{
				mWeight._m_cpu_data_address[i] = mAffineWeight * static_cast<DataType>(dist(engine));
			}
			mWeight.synchronize_from_CPU_to_GPU();
		}

		if (isBiasInit)
		{
#ifdef _DEBUG
			std::cout << "Bias Param was initialized." << std::endl;
#endif // _DEBUG
			for (u32 i = 0, end = mBias.mDataSize; i < end; i++)
			{
				mBias._m_cpu_data_address[i] = 0.0f;
			}
			mBias.synchronize_from_CPU_to_GPU();
		}
	}


	//���`������
	{
		if (m_on_cuda)
		{
			auto output_gpu_address = mOutput._m_gpu_data_address;
			auto input_gpu_address = input._m_gpu_data_address;
			auto weight_gpu_address = mWeight._m_gpu_data_address;
			auto bias_gpu_address = mBias._m_gpu_data_address;

			dim3 block(32, 32);
			dim3 grid((m_output_size + block.x - 1) / block.x, (m_batch_size + block.y - 1) / block.y);
			affine_forward_gpu_impl << <grid, block >> > (
				output_gpu_address,
				input_gpu_address,
				weight_gpu_address,
				bias_gpu_address,
				m_batch_size,
				m_output_size,
				m_input_size);
			CUDA_SYNCHRONIZE_DEBUG;
		}
		else
		{
			forward_cpu_impl(input);
		}
	}


	return iotype{ Tensor(m_output_tensorcore_tbl[0]) };
}


void AffineCore::backward()
{
	//std::cout << "Affine Backward" << std::endl;
	if (std::shared_ptr<TensorCore> input_ptr = mInputTensorCoreTbl[0].lock())
	{
		auto& input = *input_ptr;

		auto output_gpu_grad_address = mOutput._m_gpu_grad_data_address;

		auto input_gpu_address = input._m_gpu_data_address;
		auto input_gpu_grad_address = input._m_gpu_grad_data_address;

		auto weight_gpu_address = mWeight._m_gpu_data_address;
		auto weight_gpu_grad_address = mWeight._m_gpu_grad_data_address;

		auto bias_gpu_grad_address = mBias._m_gpu_grad_data_address;

		//�p�����[�^�̋t�`��
		{
			if (m_on_cuda)
			{

				//Weight
				{
					dim3 block(16, 16);
					dim3 grid((m_input_size + block.x - 1) / block.x, (m_output_size + block.y - 1) / block.y);
					affine_backward_gpu_impl_weight << <grid, block >> > (
						weight_gpu_grad_address,
						output_gpu_grad_address,
						input_gpu_address,
						m_batch_size,
						m_output_size,
						m_input_size);
					CUDA_SYNCHRONIZE_DEBUG;
				}

				//Bias
				{
					dim3 block(16);
					dim3 grid((m_output_size + block.x - 1) / block.x);
					affine_backward_gpu_impl_bias << <grid, block >> > (
						bias_gpu_grad_address,
						output_gpu_grad_address,
						m_batch_size,
						m_output_size);
					CUDA_SYNCHRONIZE_DEBUG;
				}
			}
			else
			{
				backward_cpu_impl_parameter(input_ptr);
			}
		}

		if (input.m_grad_required)//���z�s�v�̏ꍇ�A�t�`���̓X�L�b�v�o����B
		{
			if (m_on_cuda)
			{
				dim3 block(16, 16);
				dim3 grid((m_input_size + block.x - 1) / block.x, (m_batch_size + block.y - 1) / block.y);
				affine_backward_gpu_impl_input << <grid, block >> > (
					input_gpu_grad_address,
					output_gpu_grad_address,
					weight_gpu_address,
					m_batch_size,
					m_output_size,
					m_input_size);
				CUDA_SYNCHRONIZE_DEBUG;
			}
			else
			{
				backward_cpu_impl_input(input_ptr);
			}
		}
	}
	else
	{
		std::cout << "Resource0 Error@ReLUCore::backward" << std::endl;
		exit(1);
	}
}



void AffineCore::forward_cpu_impl(const TensorCore& input)
{
	for (u32 N = 0, end = mOutput.mBatchSize; N < end; N++)
	{
		for (u32 O = 0; O < m_output_size; O++)
		{
			DataType result = 0.0f;
			for (u32 I = 0; I < m_input_size; I++)
			{
				result += mWeight(O, I) * input(N, I);
			}

			mOutput(N, O) = result + mBias(O);
		}
	}
}

void AffineCore::backward_cpu_impl_input(const std::shared_ptr<TensorCore>& input_tensorcore)
{
	auto& input = *input_tensorcore;

	for (u32 N = 0; N < m_batch_size; N++)
	{
		for (u32 I = 0; I < m_input_size; I++)
		{
			DataType result = 0.0f;
			for (u32 O = 0; O < m_output_size; O++)
			{
				result += mWeight(O, I) * mOutput.d(N, O);
			}
			input.d(N, I) = result;
		}
	}
}

void AffineCore::backward_cpu_impl_parameter(const std::shared_ptr<TensorCore>& input_tensorcore)
{
	auto& input = *input_tensorcore;

	for (u32 O = 0; O < m_output_size; O++)
	{
		for (u32 I = 0; I < m_input_size; I++)
		{
			DataType result = 0.0f;
			for (u32 N = 0; N < m_batch_size; N++)
			{
				result += input(N, I) * mOutput.d(N, O);
			}
			mWeight.d(O, I) = result;
		}

		DataType result = 0.0f;
		for (u32 N = 0; N < m_batch_size; N++)
		{
			result += mOutput.d(N, O);
		}
		mBias.d(O) = result;
	}
}