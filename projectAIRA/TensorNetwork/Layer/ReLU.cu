#include "hip/hip_runtime.h"
#include "ReLU.h"
#include "nnLayer.h"



namespace
{
	void relu_forward_impl_cpu(DataType* output_ptr, DataType* input_ptr, DataType* mask_ptr, u32 data_size)
	{
		for (u32 i = 0; i < data_size; i++)
		{
			const auto& input_value = input_ptr[i];
			auto& mask = mask_ptr[i];
			if (input_value > 0)
				mask = 1;
			else
				mask = 0;
			output_ptr[i] = input_value * mask;
		}
	}

	void relu_backward_impl_cpu(DataType* d_output_ptr, DataType* d_input_ptr, DataType* mask_ptr, u32 data_size)
	{
		for (u32 i = 0; i < data_size; i++)
		{
			d_input_ptr[i] = d_output_ptr[i] * mask_ptr[i];
		}
	}


	__global__ void relu_forward_impl_gpu(DataType* output_ptr, DataType* input_ptr, DataType* mask_ptr, u32 dataSize)
	{
		u32 i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= dataSize)
		{
			return;
		}

		const DataType& input_value = input_ptr[i];
		auto& mask = mask_ptr[i];

		if (input_value > 0)
			mask = 1;
		else
			mask = 0;

		output_ptr[i] = input_value * mask;
	}


	__global__ void relu_backward_impl_gpu(DataType* doutput_ptr, DataType* dinput_ptr, DataType* mask_ptr, u32 data_size)
	{
		u32 i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= data_size)
		{
			return;
		}

		doutput_ptr[i] = dinput_ptr[i] * mask_ptr[i];
	}
}

using namespace aoba::nn::layer;
using ReLUCore = Layer::ReLUCore;
using LayerSkeleton = Layer::LayerSkeleton;

Layer::nnLayer aoba::nn::layer::ReLU()
{
	Layer::nnLayer relu = gen<ReLUCore>("Add");
	return relu;
}

ReLUCore::ReLUCore()
	: LayerSkeleton(1, 1, 1, 1)
{
}


LayerSkeleton::iotype ReLUCore::forward(const LayerSkeleton::iotype& input_tensors)
{
	const auto& input_tensorcore = *getTensorCoreFrom(input_tensors[0]);

	auto dataSize_input = input_tensorcore.mDataSize;


	//���������I����Ă��Ȃ��ꍇ�A�����ŃC���v�b�g���ꂽ�e���\���ɍ��킹���I�Ɋm��/���������s���B
	if (!m_init_finish)
	{
		auto& child_tensorcore = m_output_tensorcore_tbl[0];
		genDownStreamTensor(0, std::make_shared<TensorCore>(input_tensorcore, true));

		auto& mask = m_parameter_tbl[0];
		mask = std::make_shared<TensorCore>(input_tensorcore, false);

		if (input_tensorcore._m_on_cuda)
		{
			m_on_cuda = true;

			child_tensorcore->to_cuda("");

			//�����p�����[�^��CUDA�ɑ���B
			mask->to_cuda("");
		}
		m_init_finish = true;
	}

	auto& child_tensorcore = *m_output_tensorcore_tbl[0];
	const auto& mask = *m_parameter_tbl[0];
	auto dataSize = child_tensorcore.mDataSize;
	if (dataSize != dataSize_input)
	{
		std::cout << "Input tensor size between Input & Output is not match." << std::endl;
		exit(1);
	}


	if (m_on_cuda)
	{
		auto output_address = child_tensorcore._m_gpu_data_address;
		auto input_address = input_tensorcore._m_gpu_data_address;
		auto mask_address = mask._m_gpu_data_address;

		dim3 block(256);
		dim3 grid((dataSize + block.x - 1) / block.x);
		relu_forward_impl_gpu << <grid, block >> > (output_address, input_address, mask_address, dataSize);
		CUDA_SYNCHRONIZE_DEBUG;
	}
	else
	{
		auto output_address = child_tensorcore._m_cpu_data_address;
		auto input_address = input_tensorcore._m_cpu_data_address;
		auto mask_address = mask._m_cpu_data_address;
		relu_forward_impl_cpu(output_address, input_address, mask_address, dataSize);
	}

	return iotype{ Tensor(m_output_tensorcore_tbl[0]) };
}


void ReLUCore::backward()
{
	if (std::shared_ptr<TensorCore> input_tensor_core = mInputTensorCoreTbl[0].lock())
	{
		if (input_tensor_core->_m_need_grad)
		{
			const auto& output_tensorcore = *m_output_tensorcore_tbl[0];
			auto& input_tensorcore = *input_tensor_core;
			const auto& mask = *m_parameter_tbl[0];

			auto dataSize = output_tensorcore.mDataSize;
			if (m_on_cuda)
			{
				auto d_output_address = output_tensorcore._m_gpu_grad_data_address;
				auto d_input_address = input_tensorcore._m_gpu_grad_data_address;
				auto mask_address = mask._m_gpu_data_address;

				dim3 block(32);
				dim3 grid((dataSize + block.x - 1) / block.x);
				relu_backward_impl_gpu << <grid, block >> > (d_input_address, d_output_address, mask_address, dataSize);
				CUDA_SYNCHRONIZE_DEBUG;
				//input_tensorcore.synchronize_from_GPU_to_CPU();
			}
			else
			{
				auto output_grad_address = output_tensorcore._m_cpu_grad_data_address;
				auto input_grad_address = input_tensorcore._m_cpu_grad_data_address;
				auto mask_address = mask._m_cpu_data_address;
				relu_backward_impl_cpu(output_grad_address, input_grad_address, mask_address, dataSize);

			}
		}
	}
	else
	{
		std::cout << "Resource Error@ReLUCore::backward" << std::endl;
		exit(1);
	}
}

