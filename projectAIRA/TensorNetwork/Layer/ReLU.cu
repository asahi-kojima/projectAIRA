#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "ReLU.h"


namespace
{
	void relu_forward_impl_cpu(DataType* output_ptr, DataType* input_ptr, DataType* mask_ptr, u32 data_size)
	{
		for (u32 i = 0; i < data_size; i++)
		{
			const auto& input_value = input_ptr[i];
			auto& mask = mask_ptr[i];
			if (input_value > 0)
				mask = 1;
			else
				mask = 0;
			output_ptr[i] = input_value * mask;
		}
	}

	__global__ void relu_forward_impl_gpu(DataType* output_ptr, DataType* input_ptr, DataType* mask_ptr, u32 data_size)
	{
		u32 i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= data_size)
		{
			return;
		}

		const DataType& input_value = input_ptr[i];
		auto& mask = mask_ptr[i];

		if (input_value > 0)
			mask = 1;
		else
			mask = 0;

		output_ptr[i] = input_value * mask;
	}

	void relu_backward_impl_cpu(DataType* d_output_ptr, DataType* d_input_ptr, DataType* mask_ptr, u32 data_size)
	{
		for (u32 i = 0; i < data_size; i++)
		{
			d_input_ptr[i] = d_output_ptr[i] * mask_ptr[i];
		}
	}

	__global__ void relu_backward_impl_gpu(DataType* d_output_ptr, DataType* d_input_ptr, DataType* mask_ptr, u32 data_size)
	{
		u32 i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= data_size)
		{
			return;
		}

		d_input_ptr[i] = d_output_ptr[i] * mask_ptr[i];
	}
}

Layer ReLU()
{
	Layer relu = gen<ReLUCore>("Add");
	return relu;
}

ReLUCore::ReLUCore()
	: LayerCore(1, 1, 1)
{
	//mask�p
	m_parameter_tbl.resize(1);
}


LayerCore::iotype ReLUCore::forward(const LayerCore::iotype& input_tensors)
{
	auto dataSize_input = Accessor2TensorCore::getDataSize(input_tensors[0]);


	//���������I����Ă��Ȃ��ꍇ�A�����ŃC���v�b�g���ꂽ�e���\���ɍ��킹���I�Ɋm��/���������s���B
	if (!m_init_finish)
	{
		std::vector<u32> shape = Accessor2TensorCore::getTensorShape(input_tensors[0]);

		auto& child_tensorcore = m_child_tensorcore_tbl[0];
		child_tensorcore = std::make_shared<TensorCore>(true, shape);
		child_tensorcore->regist_parent_layercore(shared_from_this());

		auto& mask = m_parameter_tbl[0];
		mask = std::make_shared<TensorCore>(false, shape);

		if (Accessor2TensorCore::on_cuda(input_tensors[0]))
		{
			m_on_cuda = true;

			child_tensorcore->to_cuda("");

			//�����p�����[�^��CUDA�ɑ���B
			for (u32 i = 0, end = m_parameter_tbl.size(); i < end; i++)
			{
				m_parameter_tbl[i]->to_cuda("");
			}
		}
		m_init_finish = true;
	}

	auto dataSize = Accessor2TensorCore::getDataSize(m_child_tensorcore_tbl[0]);
	if (dataSize != dataSize_input)
	{
		std::cout << "Input tensor size between Input & Output is not match." << std::endl;
		exit(1);
	}



	std::cout << "ReLU forward " << (m_on_cuda ? "On GPU" : "on CPU") << std::endl;
	if (m_on_cuda)
	{
		auto output_address = Accessor2TensorCore::getAddressOnGpuFrom(m_child_tensorcore_tbl[0]);
		auto input_address = Accessor2TensorCore::getAddressOnGpuFrom(input_tensors[0]);
		auto mask_address = Accessor2TensorCore::getAddressOnGpuFrom(m_parameter_tbl[0]);

		dim3 block(256);
		dim3 grid((dataSize + block.x - 1) / block.x);
		relu_forward_impl_gpu << <grid, block >> > (output_address, input_address, mask_address, dataSize);
		CHECK(hipDeviceSynchronize());
	}
	else
	{
		auto output_address = Accessor2TensorCore::getAddressOnCpuFrom(m_child_tensorcore_tbl[0]);
		auto input_address = Accessor2TensorCore::getAddressOnCpuFrom(input_tensors[0]);
		auto mask_address = Accessor2TensorCore::getAddressOnCpuFrom(m_parameter_tbl[0]);
		relu_forward_impl_cpu(output_address, input_address, mask_address, dataSize);
	}

	return iotype{ Tensor(m_child_tensorcore_tbl[0]) };
}

void ReLUCore::backward()
{
	std::cout << "ReLU backward" << std::endl;
	if (std::shared_ptr<TensorCore> input_tensor_core = mInputTensorCoreTbl[0].lock())
	{
		if (Accessor2TensorCore::get_need_grad(input_tensor_core))
		{

			auto dataSize = Accessor2TensorCore::getDataSize(m_child_tensorcore_tbl[0]);
			if (m_on_cuda)
			{
				auto output_address = Accessor2TensorCore::getGradAddressOnGpuFrom(m_child_tensorcore_tbl[0]);
				auto input_address = Accessor2TensorCore::getGradAddressOnGpuFrom(input_tensor_core);
				auto mask_address = Accessor2TensorCore::getAddressOnGpuFrom(m_parameter_tbl[0]);

				dim3 block(256);
				dim3 grid((dataSize + block.x - 1) / block.x);
				relu_backward_impl_gpu << <grid, block >> > (output_address, input_address, mask_address, dataSize);
				CHECK(hipDeviceSynchronize());

			}
			else
			{
				auto output_grad_address = Accessor2TensorCore::getGradAddressOnCpuFrom(m_child_tensorcore_tbl[0]);
				auto input_grad_address = Accessor2TensorCore::getGradAddressOnCpuFrom(input_tensor_core);
				auto mask_address = Accessor2TensorCore::getAddressOnCpuFrom(m_parameter_tbl[0]);
				relu_backward_impl_cpu(output_grad_address, input_grad_address, mask_address, dataSize);

			}
		}
	}
	else
	{
		std::cout << "Resource Error@ReLUCore::backward" << std::endl;
		exit(1);
	}
}