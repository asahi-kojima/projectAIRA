#include "hip/hip_runtime.h"
#include "ReLU.h"
#include "Layer.h"



namespace
{
	__global__ void forward_impl_gpu(DataType* output_ptr, const DataType* input_ptr, DataType* mask_ptr, u32 dataSize)
	{
		u32 i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= dataSize)
		{
			return;
		}

		const DataType& input_value = input_ptr[i];
		auto& mask = mask_ptr[i];

		if (input_value > 0)
			mask = 1;
		else
			mask = 0;

		output_ptr[i] = input_value * mask;
	}


	__global__ void backward_impl_gpu(DataType* doutput_ptr, const DataType* dinput_ptr, const DataType* mask_ptr, u32 data_size)
	{
		u32 i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= data_size)
		{
			return;
		}

		doutput_ptr[i] = dinput_ptr[i] * mask_ptr[i];
	}
}


namespace aoba
{
	namespace nn
	{
		namespace layer
		{

			Layer aoba::nn::layer::ReLU()
			{
				Layer relu = gen<ReLUCore>("Add");
				return relu;
			}

			ReLUCore::ReLUCore()
				: BaseLayer(1, 1, 1)
				, mDataSize(0)
				, mOutput(*m_output_tensorcore_tbl[0])
				, mMask(false)
			{
			}


			BaseLayer::iotype ReLUCore::forward(const BaseLayer::iotype& input_tensors)
			{
				if (!m_init_finish)
				{
					initialize();
				}

				const auto& input = *getTensorCoreFrom(input_tensors[0]);



				//�o�̓e���\���ƃp�����[�^�̌`��m�F���Ή�
				{
					//�f�[�^�T�C�Y���i�[
					mDataSize = input.getDataSize();
					//m_on_cuda = input.m_on_cuda;

					//�o�̓e���\���̌`��ύX
					mOutput.reshapeAs(input, input.isOnCuda());

					//�}�X�N�̌`��ύX
					mMask.reshapeAs(input, input.isOnCuda());
				}




				if (m_on_cuda)
				{
					auto output_gpu_address = mOutput.getGpuDataAddress();
					auto input_gpu_address = input.getGpuDataAddress();
					auto mask_gpu_address = mMask.getGpuDataAddress();

					dim3 block(256);
					dim3 grid((mDataSize + block.x - 1) / block.x);
#ifdef TIME_DEBUG
					std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
#endif // TIME_DEBUG
					forward_impl_gpu << <grid, block >> > (output_gpu_address, input_gpu_address, mask_gpu_address, mDataSize);
					CUDA_SYNCHRONIZE_DEBUG;
#ifdef TIME_DEBUG
					f32 elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start).count() / 1000.0f;
					std::string name = makeDebugIdentifier(mInstanceID, __FUNCTION__, "forward_impl_gpu");
					debugTimers[name] = elapsedTime;
#endif // TIME_DEBUG
				}
				else
				{
#ifdef TIME_DEBUG
					std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
#endif // TIME_DEBUG
					forward_cpu_impl(input);
#ifdef TIME_DEBUG
					f32 elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start).count() / 1000.0f;
					std::string name = makeDebugIdentifier(mInstanceID, __FUNCTION__, "forward_cpu_impl");
					debugTimers[name] = elapsedTime;
#endif // TIME_DEBUG
				}

				return iotype{ Tensor(m_output_tensorcore_tbl[0]) };
			}


			void ReLUCore::backward()
			{
				if (const std::shared_ptr<TensorCore>& input_tensor_core = mInputTensorCoreTbl[0].lock())
				{
					TensorCore& input = *input_tensor_core;
					if (input.requiresGrad())
					{
						auto& input_tensorcore = *input_tensor_core;
						if (m_on_cuda)
						{
							auto output_gpu_grad_address = mOutput.getGpuGradDataAddress();
							auto input_gpu_grad_address = input_tensorcore.getGpuGradDataAddress();
							auto mask_gpu_address = mMask.getGpuDataAddress();

							dim3 block(32);
							dim3 grid((mDataSize + block.x - 1) / block.x);
#ifdef TIME_DEBUG
							std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
#endif // TIME_DEBUG
							backward_impl_gpu << <grid, block >> > (input_gpu_grad_address, output_gpu_grad_address, mask_gpu_address, mDataSize);
							CUDA_SYNCHRONIZE_DEBUG;
#ifdef TIME_DEBUG
							f32 elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start).count() / 1000.0f;
							std::string name = makeDebugIdentifier(mInstanceID, __FUNCTION__, "backward_impl_gpu");
							debugTimers[name] = elapsedTime;
#endif // TIME_DEBUG
						}
						else
						{
#ifdef TIME_DEBUG
							std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
#endif // TIME_DEBUG
							backward_cpu_impl(*input_tensor_core);
#ifdef TIME_DEBUG
							f32 elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start).count() / 1000.0f;
							std::string name = makeDebugIdentifier(mInstanceID, __FUNCTION__, "backward_cpu_impl");
							debugTimers[name] = elapsedTime;
#endif // TIME_DEBUG
						}
					}
				}
				else
				{
					std::cout << "Resource Error@ReLUCore::backward" << std::endl;
					exit(1);
				}
			}


			void ReLUCore::forward_cpu_impl(const TensorCore& input)
			{
				for (u32 i = 0; i < mDataSize; i++)
				{
					const auto& input_value = input[i];
					auto& mask = mMask[i];

					if (input_value > 0)
						mask = 1;
					else
						mask = 0;

					mOutput[i] = input_value * mask;
				}
			}
			void ReLUCore::backward_cpu_impl(TensorCore& input)
			{
				for (u32 i = 0; i < mDataSize; i++)
				{
					input.d(i) = mOutput.d(i) * mMask[i];
				}
			}

		}
	}
}