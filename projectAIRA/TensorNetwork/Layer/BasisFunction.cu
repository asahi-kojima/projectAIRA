#include "hip/hip_runtime.h"
#include "BasisFunction.h"


namespace
{
	//Tanh�֌W
	__host__ __device__ DataType hd_tanh_forward(DataType x)
	{
		return tanh(x);
	}
	__global__ void g_tanh_forward(DataType* output, const DataType* input, u32 dataSize)
	{
		const u32 index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= dataSize)
		{
			return;
		}

		output[index] = hd_tanh_forward(input[index]);
	}

	__host__ __device__ DataType hd_tanh_backward(DataType output_grad, DataType input)
	{
		const DataType cosh_value = cosh(input);
		return output_grad * (1 / (cosh_value * cosh_value));
	}
	__global__ void g_tanh_backward(DataType* input_grad, const DataType* output_grad, const DataType* input, u32 dataSize)
	{
		const u32 index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= dataSize)
		{
			return;
		}

		const DataType output_grad_value = output_grad[index];
		const DataType input_value = input[index];
		input_grad[index] = hd_tanh_backward(output_grad_value, input_value);
	}


	//Sigmoid�֌W
	__host__ __device__ DataType hd_sigmoid_forward(DataType x)
	{
		return 1 / (1 + exp(-x));
	}
	__global__ void g_sigmoid_forward(DataType* output, const DataType* input, u32 dataSize)
	{
		const u32 index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= dataSize)
		{
			return;
		}

		output[index] = hd_sigmoid_forward(input[index]);
	}

	__host__ __device__ DataType hd_sigmoid_backward(DataType output_grad, DataType input)
	{
		const DataType exp_ninus_x = exp(-input);
		return output_grad * (exp_ninus_x / ((1+ exp_ninus_x) * (1+ exp_ninus_x)));
	}
	__global__ void g_sigmoid_backward(DataType* input_grad, const DataType* output_grad, const DataType* input, u32 dataSize)
	{
		const u32 index = blockIdx.x * blockDim.x + threadIdx.x;
		if (index >= dataSize)
		{
			return;
		}

		const DataType output_grad_value = output_grad[index];
		const DataType input_value = input[index];
		input_grad[index] = hd_sigmoid_backward(output_grad_value, input_value);
	}
}

namespace aoba
{
	namespace nn
	{
		namespace layer
		{
			Layer Tanh()
			{
				Layer tanh = gen<BasisFunctionCore>("Tanh", hd_tanh_forward, g_tanh_forward, hd_tanh_backward, g_tanh_backward);
				return tanh;
			}

			Layer Sigmoid()
			{
				Layer sigmoid = gen<BasisFunctionCore>("Sigmoid", hd_sigmoid_forward, g_sigmoid_forward, hd_sigmoid_backward, g_sigmoid_backward);
				return sigmoid;
			}



			BasisFunctionCore::BasisFunctionCore(
				CPUFunctionF functionCPU_forward, GPUFunctionF functionGPU_forward,
				CPUFunctionB functionCPU_backward, GPUFunctionB functionGPU_backward)
				:BaseLayer(1,1,1)
				,mOutput(*m_output_tensorcore_tbl[0])
				,mDataSize(0)
				,mFunctionCPU_forward(functionCPU_forward)
				,mFunctionGPU_forward(functionGPU_forward)
				,mFunctionCPU_backward(functionCPU_backward)
				,mFunctionGPU_backward(functionGPU_backward)
			{

			}

			

			BasisFunctionCore::~BasisFunctionCore()
			{

			}

			BaseLayer::iotype BasisFunctionCore::forward(const iotype& input_tensors)
			{
				if (!m_init_finish)
				{
					initialize();
				}

				const auto& input = *getTensorCoreFrom(input_tensors[0]);

				{
					mDataSize = input.getDataSize();

					mOutput.reshapeAs(input, m_on_cuda);
				}


				if (m_on_cuda)
				{
					auto output_gpu_address = mOutput.getGpuDataAddress();
					auto input_gpu_address = input.getGpuDataAddress();
					dim3 block(128);
					dim3 grid((mDataSize + block.x - 1) / block.x);
#ifdef TIME_DEBUG
					std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
#endif // TIME_DEBUG
					mFunctionGPU_forward << <grid, block >> > (output_gpu_address, input_gpu_address, mDataSize);
					CUDA_SYNCHRONIZE_DEBUG;
#ifdef TIME_DEBUG
					f32 elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start).count() / 1000.0f;
					std::string name = makeDebugIdentifier(mInstanceID, __FUNCTION__, "mFunctionGPU_forward");
					debugTimers[name] = elapsedTime;
#endif // TIME_DEBUG
				}
				else
				{
#ifdef TIME_DEBUG
					std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
#endif // TIME_DEBUG
					forward_cpu_impl(input);
#ifdef TIME_DEBUG
					f32 elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start).count() / 1000.0f;
					std::string name = makeDebugIdentifier(mInstanceID, __FUNCTION__, "forward_cpu_impl");
					debugTimers[name] = elapsedTime;
#endif // TIME_DEBUG
				}


				return iotype{ m_output_tensorcore_tbl[0] };
			}

			void BasisFunctionCore::backward()
			{
				if (const std::shared_ptr<TensorCore>& input_ptr = mInputTensorCoreTbl[0].lock())
				{
					TensorCore& input = *input_ptr;
					if (input.requiresGrad())
					{
						if (m_on_cuda)
						{
							auto input_gpu_grad_address = input.getGpuGradDataAddress();
							const auto output_gpu_grad_address = mOutput.getGpuGradDataAddress();
							const auto input_gpu_address = input.getGpuDataAddress();

							dim3 block(32);
							dim3 grid((mDataSize + block.x - 1) / block.x);
#ifdef TIME_DEBUG
							std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
#endif // TIME_DEBUG
							mFunctionGPU_backward << <grid, block >> > (
								input_gpu_grad_address, 
								output_gpu_grad_address,
								input_gpu_address,
								mDataSize);
							CUDA_SYNCHRONIZE_DEBUG;
#ifdef TIME_DEBUG
							f32 elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start).count() / 1000.0f;
							std::string name = makeDebugIdentifier(mInstanceID, __FUNCTION__, "mFunctionGPU_backward");
							debugTimers[name] = elapsedTime;
#endif // TIME_DEBUG
						}
						else
						{
#ifdef TIME_DEBUG
							std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
#endif // TIME_DEBUG
							backward_cpu_impl(input);
#ifdef TIME_DEBUG
							f32 elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start).count() / 1000.0f;
							std::string name = makeDebugIdentifier(mInstanceID, __FUNCTION__, "backward_cpu_impl");
							debugTimers[name] = elapsedTime;
#endif // TIME_DEBUG
						}
					}
				}
				else
				{
					std::cout << "Resource Error@ReLUCore::backward" << std::endl;
					exit(1);
				}
			}

			void BasisFunctionCore::forward_cpu_impl(const TensorCore& input)
			{
				for (u32 i = 0; i < mDataSize; i++)
				{
					const auto input_value = input[i];
					mOutput[i] = mFunctionCPU_forward(input_value);
				}
			}
			void BasisFunctionCore::backward_cpu_impl(TensorCore& input)
			{
				for (u32 i = 0; i < mDataSize; i++)
				{
					const auto output_grad_value = mOutput.d(i);
					const auto input_value = input(i);
					input.d(i) = mFunctionCPU_backward(output_grad_value, input_value);
				}
			}
		}
	}
}